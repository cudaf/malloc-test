#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "support.h"


// Testing performance of 100 memory copy operations
// between CPU memory allocated with malloc().
float test_malloc(int size) {
  void *a = malloc(size);
  void *b = malloc(size);
  clock_t start = clock();

  for (int i=0; i<100; i++)
    memcpy(b, a, size);

  clock_t stop = clock();
  float duration = (float)(stop - start) / CLOCKS_PER_SEC;
  return duration * 1000;
}


// Testing performance of 100 memory copy operations
// between CPU memory allocated with malloc() and
// GPU memory allocated with hipMalloc(). Because
// memory allocated with malloc() is pageable memory,
// it will first be copied to a page-locked `staging`
// area, before being transferring to GPU by DMA.
// Note however that allocating too much pinned memory
// can cause system slowdown, or even crash due to
// lack of usable memory.
float test_cuda_malloc(int size, bool up) {
  hipEvent_t start, stop;
  TRY( hipEventCreate(&start) );
  TRY( hipEventCreate(&stop) );

  void *a, *aD;
  a = malloc(size);
  TRY( hipMalloc(&aD, size) );
  TRY( hipEventRecord(start, 0) );

  for (int i=0; i<100; i++) {
    if (up) TRY( hipMemcpy(aD, a, size, hipMemcpyHostToDevice) );
    else TRY( hipMemcpy(a, aD, size, hipMemcpyDeviceToHost) );
  }

  float duration;
  TRY( hipEventRecord(stop, 0) );
  TRY( hipEventSynchronize(stop) );
  TRY( hipEventElapsedTime(&duration, start, stop) );

  TRY( hipEventDestroy(start) );
  TRY( hipEventDestroy(stop) );
  TRY( hipFree(aD) );
  free(a);
  return duration;
}


// Testing performance of 100 memory copy operations
// between CPU memory allocated with hipHostAlloc()
// and GPU memory allocated with hipMalloc(). Memory
// allocated with hipHostAlloc() is page-locked
// (pinned), which means the memory can be directly
// copied by DMA into the GPU.
float test_cuda_host_alloc(int size, bool up) {
  hipEvent_t start, stop;
  TRY( hipEventCreate(&start) );
  TRY( hipEventCreate(&stop) );

  void *a, *aD;
  TRY( hipHostAlloc(&a, size, hipHostMallocDefault) );
  TRY( hipMalloc(&aD, size) );
  TRY( hipEventRecord(start, 0) );

  for (int i=0; i<100; i++) {
    if (up) TRY( hipMemcpy(aD, a, size, hipMemcpyHostToDevice) );
    else TRY( hipMemcpy(a, aD, size, hipMemcpyDeviceToHost) );
  }

  float duration;
  TRY( hipEventRecord(stop, 0) );
  TRY( hipEventSynchronize(stop) );
  TRY( hipEventElapsedTime(&duration, start, stop) );

  TRY( hipEventDestroy(start) );
  TRY( hipEventDestroy(stop) );
  TRY( hipFree(aD) );
  TRY( hipHostFree(a) );
  return duration;
}


int main() {
  int size = 10 * 1024 * 1024;

  printf("CPU malloc -> CPU malloc: %3.1f ms\n",
    test_malloc(size));
  printf("\n");

  printf("CPU malloc -> GPU hipMalloc: %3.1f ms\n",
    test_cuda_malloc(size, 1));
  printf("CPU malloc <- GPU hipMalloc: %3.1f ms\n",
    test_cuda_malloc(size, 0));
  printf("\n");

  printf("CPU hipHostAlloc -> GPU hipMalloc: %3.1f ms\n",
    test_cuda_host_alloc(size, 1));
  printf("CPU hipHostAlloc <- GPU hipMalloc: %3.1f ms\n",
    test_cuda_host_alloc(size, 0));
  return 0;
}
