#include <cstdio>
#include "src/main.hxx"

using std::printf;


int main() {
  int size = 10 * 1024 * 1024;

  printf("CPU malloc -> CPU malloc: %3.1f ms\n",
    testMalloc(size));
  printf("\n");

  printf("CPU malloc -> GPU hipMalloc: %3.1f ms\n",
    testCudaMalloc(size, 1));
  printf("CPU malloc <- GPU hipMalloc: %3.1f ms\n",
    testCudaMalloc(size, 0));
  printf("\n");

  printf("CPU hipHostAlloc -> GPU hipMalloc: %3.1f ms\n",
    testCudaHostAlloc(size, 1));
  printf("CPU hipHostAlloc <- GPU hipMalloc: %3.1f ms\n",
    testCudaHostAlloc(size, 0));
  printf("\n");
  return 0;
}
